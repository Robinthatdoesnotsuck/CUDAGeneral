#include "hip/hip_runtime.h"


#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void cuda_hello() {
    // Instructions have to be in C in GPU instructions
    printf("Hello World from GPU!\n");
}

int main() {
    // blocks in a grid, number of threads in a block\
    // grid, block
    cuda_hello <<<1, 1>>> ();
    hipDeviceSynchronize(); // Wait for the GPU to finish
    cout << "Hello World from CPU!" << endl;
    return 0;
}
